#include "hip/hip_runtime.h"
#include "board.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

// nvcc test.cu -O2 -o test
// srun --reservation=fri -G1 -n1 test

// Grid dimensions

__global__ void updateGrid(unsigned int* d_board, unsigned int* d_board_new, float* d_levels, float* d_levels_new, int n_rows, int n_cols, float alpha, float gamma)
{

    // precompute coefficients
    float alpha12 = alpha / 12.;
    float alpha2 = alpha / 2.;
    char neighs_even_col[6][2] = { { -1, 0 }, { -1, 1 }, { 0, 1 }, { 1, 0 }, { 0, -1 }, { -1, -1 } };
    char neighs_odd_col[6][2] = { { -1, 0 }, { 0, 1 }, { 1, 1 }, { 1, 0 }, { 1, -1 }, { 0, -1 } };
    char(*neighs)[2]; // the pointer to the neighbor offset arrays

    int col = blockIdx.x * blockDim.x + threadIdx.x; // column
    int row = blockIdx.y * blockDim.y + threadIdx.y; // row

    int idx = row * n_cols + col;

    // Check if the coordinates are within the grid bounds
    // and current cell is not frozen
    // 1,2,3 cells receive water
    if (col < n_cols && row < n_rows && d_board[idx] != 0) {
        neighs = (col % 2 == 0) ? neighs_even_col : neighs_odd_col;

        // find the accumulation of water from the neighbors
        for (int k = 0; k < 6; k++) {
            int neigh_row = row + neighs[k][0]; // y
            int neigh_col = col + neighs[k][1]; // x

            // neighboring edge and unreceptive cells contributes water
            if (d_board[neigh_row * n_cols + neigh_col] < 2) {
                d_levels_new[idx] += alpha12 * d_levels[neigh_row * n_cols + neigh_col];
            }
        }

        // if the cell is boundary
        if (d_board[idx] == 2) {
            d_levels_new[idx] += gamma; // add the water vapor

            // if the cell freezes, set the flag and update the neighbor cells
            if (d_levels_new[idx] >= 1) {
                d_board_new[idx] = 3;

                for (int k = 0; k < 6; k++) {
                    int neigh_row = row + neighs[k][0];
                    int neigh_col = col + neighs[k][1];

                    // we dont update edge cells
                    if (d_board[neigh_row * n_cols + neigh_col] != 0) {
                        atomicMax(&d_board_new[neigh_row * n_cols + neigh_col], 2);
                    }
                }
            }

        } else {
            // otherwise the water also difuses out of the cell
            d_levels_new[idx] -= alpha2 * d_levels[idx];
        }
    }
}

void saveBoardToFile(unsigned int* board, unsigned int n, unsigned int m, const char* filename)
{
    FILE* fp = fopen(filename, "wb+");
    if (fp != NULL) {
        fwrite(board, sizeof(unsigned int), n * m, fp);
        fclose(fp);
        printf("Board saved to file: %s\n", filename);
    } else {
        printf("Failed to open file for writing: %s\n", filename);
    }
}

void saveLevelsToFile(float* levels, unsigned int n, unsigned int m, const char* filename)
{
    FILE* fp = fopen(filename, "wb+");
    if (fp != NULL) {
        fwrite(levels, sizeof(float), n * m, fp);
        fclose(fp);
        printf("Levels saved to file: %s\n", filename);
    } else {
        printf("Failed to open file for writing: %s\n", filename);
    }
}

int main(int argc, char* argv[])
{

    int n = atoi(argv[1]);
    int m = atoi(argv[2]);
    float alpha = atof(argv[3]);
    float beta = atof(argv[4]);
    float gamma = atof(argv[5]);
    int write_to_file = atoi(argv[6]);
    int block_size_x = atoi(argv[7]);
    int block_size_y = atoi(argv[8]);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    printf("Initializing board..\n");

    // n rows and m columns
    unsigned int** board = board_initialize(n, m);
    unsigned int** board_new = board_initialize(n, m);
    float** levels = levels_initialize(n, m, beta);
    float** levels_new = levels_initialize(n, m, beta);

    // Step 1: Allocate device memory for the arrays
    // 0 -> edge cell
    // 1 -> unreceptive
    // 2 -> boundary
    // 3 -> frozen
    unsigned int* d_board;
    unsigned int* d_board_new;
    float* d_levels;
    float* d_levels_new;

    hipEventRecord(start);

    hipMalloc((void**)&d_board, n * m * sizeof(unsigned int));
    hipMalloc((void**)&d_board_new, n * m * sizeof(unsigned int));
    hipMalloc((void**)&d_levels, n * m * sizeof(float));
    hipMalloc((void**)&d_levels_new, n * m * sizeof(float));

    hipMemcpy(d_board, board[0], n * m * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_board_new, board_new[0], n * m * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_levels, levels[0], n * m * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_levels_new, levels_new[0], n * m * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float copyTime;
    hipEventElapsedTime(&copyTime, start, stop);

    int numBlocksX = (m + block_size_x - 1) / block_size_x;
    int numBlocksY = (n + block_size_y - 1) / block_size_y;

    // printf("numBlocksX: %d\n", numBlocksX);
    // printf("numBlocksY: %d\n", numBlocksY);

    // Set grid and block dimensions
    dim3 gridSize(numBlocksX, numBlocksY);
    dim3 blockSize(block_size_x, block_size_y);

    float allKernelTime = 0;
    float allWriteTime = 0;
    for (int iter = 0; iter < n / 2; iter++) {

        hipEventRecord(start);
        // Launch the kernel to update the grid
        updateGrid<<<gridSize, blockSize>>>(d_board, d_board_new, d_levels, d_levels_new, n, m, alpha, gamma);

        // Copy data from d_board_new to d_board on the GPU
        hipMemcpy(d_board, d_board_new, n * m * sizeof(unsigned int), hipMemcpyDeviceToDevice);
        hipMemcpy(d_levels, d_levels_new, n * m * sizeof(unsigned int), hipMemcpyDeviceToDevice);

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float kernelTime;
        hipEventElapsedTime(&kernelTime, start, stop);
        allKernelTime += kernelTime;

        if (write_to_file) {
            hipEventRecord(start);
            unsigned int* board_result = new unsigned int[n * m];
            hipMemcpy(board_result, d_board, n * m * sizeof(unsigned int), hipMemcpyDeviceToHost);
            float* levels_result = new float[n * m];
            hipMemcpy(levels_result, d_levels, n * m * sizeof(float), hipMemcpyDeviceToHost);

            // Save the board to a file
            char filename[50];
            snprintf(filename, sizeof(filename), "Data/array_%d.bin", iter);
            saveBoardToFile(board_result, n, m, filename);

            // Save the levels to a file
            snprintf(filename, sizeof(filename), "Data/levels_%d.bin", iter);
            saveLevelsToFile(levels_result, n, m, filename);

            hipEventRecord(stop);
            hipEventSynchronize(stop);

            float writeTime;
            hipEventElapsedTime(&writeTime, start, stop);
            allWriteTime += writeTime;
        }
    }

    hipFree(d_board);
    hipFree(d_board_new);
    hipFree(d_levels);
    hipFree(d_levels_new);

    printf("Copy time: %f ms\n", copyTime);
    printf("Kernel time: %f ms\n", allKernelTime);
    printf("Write time: %f ms\n", allWriteTime);

    return 0;
}